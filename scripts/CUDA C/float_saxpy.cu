
#include <hip/hip_runtime.h>
#include <stdio.h>

#define M 65536
#define K 128

// Implementation of SAXPY taken from Lecture Notes
__global__ void saxpy(float A[M], float B[M]) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    A[i] = 2.0 * A[i] + B[i];
}

void verifySaxpy(float A[M], int check) {
    int maxError = 0;
    
    for (int i = 0; i < M; i++) {
        maxError += abs(A[i] - check);
    }
    
    printf("Maximum Error = %d\n", maxError);
}

float saxpyHost(bool verbose) {
    float A[M], B[M];
    
    for (int i = 0; i < M; i++) {
        A[i] = 1.0f;
        B[i] = 2.0f;
    }
    
    float *dA, *dB;
    hipMalloc((void**)&dA, sizeof(float) * M);
    hipMalloc((void**)&dB, sizeof(float) * M);
    hipMemcpy(dA, A, sizeof(float) * M, hipMemcpyHostToDevice);
    hipMemcpy(dB, B, sizeof(float) * M, hipMemcpyHostToDevice);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    saxpy<<<M/K, K>>>(dA, dB);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    hipMemcpy(A, dA, sizeof(float) * M, hipMemcpyDeviceToHost);
    hipFree(dA);
    hipFree(dB);
    
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    
    if (verbose) {
        printf("Elapsed Time = %f milliseconds\n", elapsedTime);
        verifySaxpy(A, 4);
    }
    
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return elapsedTime;
}

int main() {
	int count = 100;
	float averageTime = 0;

	for (int i = 0; i < count; i++)
		averageTime += saxpyHost(false);

	averageTime /= count;
	printf("[GPU - Float] (SAXPY) Average Elapsed Time = %f ms\n", averageTime);
	return 0;
}
