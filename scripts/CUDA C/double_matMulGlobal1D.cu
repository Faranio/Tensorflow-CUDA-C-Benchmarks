
#include <hip/hip_runtime.h>
#include <stdio.h>

#define M 512
#define P 256
#define N 128
#define K 64

// My own implementation of the Matrix Multiplication
__global__ void globalMatrixMultiplication1D(double A[M][P], double B[P][N], double C[M][N]) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int i = (int)(idx / N);
    int j = idx % N;
    
    for (int k = 0; k < P; k++)
        C[i][j] += A[i][k] * B[k][j];
}

void verifyMatrixMultiplication(double C[M][N], double check) {
    int maxError = 0;
    
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            maxError += (int)abs(C[i][j] - check);
        }
    }
    
    printf("Maximum Error = %d\n", maxError);
}

float matrixMultiplication1DHost(bool verbose) {
    double A[M][P], B[P][N], C[M][N];
    
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < P; j++)
            A[i][j] = 1.0;
    }
    
    for (int i = 0; i < P; i++) {
        for (int j = 0; j < N; j++)
            B[i][j] = 2.0;
    }

    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++)
            C[i][j] = 0.0;
    }
    
    double (*dA)[P], (*dB)[N], (*dC)[N];
    hipMalloc((void**)&dA, sizeof(double) * M * P);
    hipMalloc((void**)&dB, sizeof(double) * P * N);
    hipMalloc((void**)&dC, sizeof(double) * M * N);
    hipMemcpy(dA, A, sizeof(double) * M * P, hipMemcpyHostToDevice);
    hipMemcpy(dB, B, sizeof(double) * P * N, hipMemcpyHostToDevice);
    hipMemcpy(dC, C, sizeof(double) * M * N, hipMemcpyHostToDevice);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    
    globalMatrixMultiplication1D<<<M*N/K, K>>>(dA, dB, dC);
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    hipMemcpy(C, dC, sizeof(double) * M * N, hipMemcpyDeviceToHost);
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    
    if (verbose) {
        printf("Elapsed Time = %f milliseconds\n", elapsedTime);
        verifyMatrixMultiplication(C, P * 1.0 * 2.0);
    }
    
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return elapsedTime;
}

int main() {
	int count = 100;
	float averageTime = 0;

	for (int i = 0; i < count; i++)
		averageTime += matrixMultiplication1DHost(false);

	averageTime /= count;
	printf("[GPU - Double] (Matrix Multiplication 1D - Global) Average Elapsed Time = %f ms\n", averageTime);
	return 0;
}
