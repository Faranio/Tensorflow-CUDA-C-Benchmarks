
#include <hip/hip_runtime.h>
#include <stdio.h>

#define M 512
#define K 128

// Implementation of the Matrix Add taken from the Lecture Notes
__global__ void matrixAdd1D(float A[M][M], float B[M][M]) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int i = (int)(idx / M);
    int j = idx % M;
    A[i][j] += B[i][j];
}

void verifyMatrixAdd(float A[M][M], int check) {
    int maxError = 0;
    
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < M; j++) {
            maxError += (int)abs(A[i][j] - check);
        }
    }
    
    printf("Maximum Error = %d\n", maxError);
}

float matrixAddHost(bool verbose) {
    float A[M][M], B[M][M];
    
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < M; j++) {
            A[i][j] = 1.0;
            B[i][j] = 2.0;
        }
    }
    
    float (*dA)[M], (*dB)[M];
    hipMalloc((void**)&dA, sizeof(float) * M * M);
    hipMalloc((void**)&dB, sizeof(float) * M * M);
    hipMemcpy(dA, A, sizeof(float) * M * M, hipMemcpyHostToDevice);
    hipMemcpy(dB, B, sizeof(float) * M * M, hipMemcpyHostToDevice);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    
    matrixAdd1D<<<M*M/K, K>>>(dA, dB);
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    hipMemcpy(A, dA, sizeof(float) * M * M, hipMemcpyDeviceToHost);
    hipFree(dA);
    hipFree(dB);
    
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    
    if (verbose) {
        printf("Elapsed Time = %f milliseconds\n", elapsedTime);
        verifyMatrixAdd(A, 3.0);
    }
    
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return elapsedTime;
}

int main() {
	int count = 100;
	float averageTime = 0;

	for (int i = 0; i < count; i++)
		averageTime += matrixAddHost(false);

	averageTime /= count;
	printf("[GPU - Float] (Matrix Add 1D) Average Elapsed Time = %f ms\n", averageTime);
	return 0;
}
