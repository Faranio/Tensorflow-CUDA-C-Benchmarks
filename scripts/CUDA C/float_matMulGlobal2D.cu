
#include <hip/hip_runtime.h>
#include <stdio.h>

#define M 512
#define P 256
#define N 128
#define K 8

// Implementation of the Matrix Multiplication taken from the Lecture Notes
__global__ void globalMatrixMultiplication(float A[M][P], float B[P][N], float C[M][N]) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    
    for (int k = 0; k < P; k++)
        C[i][j] += A[i][k] * B[k][j];
}

void verifyMatrixMultiplication(float C[M][N], float check) {
    int maxError = 0;
    
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            maxError += (int)abs(C[i][j] - check);
        }
    }
    
    printf("Maximum Error = %d\n", maxError);
}

float matrixMultiplicationHost(bool verbose) {
    float A[M][P], B[P][N], C[M][N];
    
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < P; j++)
            A[i][j] = 1.0f;
    }
    
    for (int i = 0; i < P; i++) {
        for (int j = 0; j < N; j++)
            B[i][j] = 2.0f;
    }

    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++)
            C[i][j] = 0.0f;
    }
    
    float (*dA)[P], (*dB)[N], (*dC)[N];
    hipMalloc((void**)&dA, sizeof(float) * M * P);
    hipMalloc((void**)&dB, sizeof(float) * P * N);
    hipMalloc((void**)&dC, sizeof(float) * M * N);
    hipMemcpy(dA, A, sizeof(float) * M * P, hipMemcpyHostToDevice);
    hipMemcpy(dB, B, sizeof(float) * P * N, hipMemcpyHostToDevice);
    hipMemcpy(dC, C, sizeof(float) * M * N, hipMemcpyHostToDevice);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    dim3 threadPerBlocks(K, K);
    dim3 numBlocks(M/K, N/K);
    
    hipEventRecord(start);
    
    globalMatrixMultiplication<<<numBlocks, threadPerBlocks>>>(dA, dB, dC);
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    hipMemcpy(C, dC, sizeof(float) * M * N, hipMemcpyDeviceToHost);
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    
    if (verbose) {
        printf("Elapsed Time = %f milliseconds\n", elapsedTime);
        verifyMatrixMultiplication(C, P * 1.0 * 2.0);
    }
    
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return elapsedTime;
}

int main() {
	int count = 100;
	float averageTime = 0;

	for (int i = 0; i < count; i++)
		averageTime += matrixMultiplicationHost(false);

	averageTime /= count;
	printf("[GPU - Float] (Matrix Multiplication 2D - Global) Average Elapsed Time = %f ms\n", averageTime);
	return 0;
}
