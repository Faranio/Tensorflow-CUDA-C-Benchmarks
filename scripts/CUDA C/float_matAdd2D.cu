
#include <hip/hip_runtime.h>
#include <stdio.h>

#define M 512
#define K 8

// Implementation of the Matrix Add taken from the Lecture Notes
__global__ void matrixAdd(float A[M][M], float B[M][M]) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    A[i][j] += B[i][j];
}

void verifyMatrixAdd(float A[M][M], int check) {
    int maxError = 0;
    
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < M; j++) {
            maxError += (int)abs(A[i][j] - check);
        }
    }
    
    printf("Maximum Error = %d\n", maxError);
}

float matrixAddHost(bool verbose) {
    float A[M][M], B[M][M];
    
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < M; j++) {
            A[i][j] = 1.0f;
            B[i][j] = 2.0f;
        }
    }
    
    float (*dA)[M], (*dB)[M];
    hipMalloc((void**)&dA, sizeof(float) * M * M);
    hipMalloc((void**)&dB, sizeof(float) * M * M);
    hipMemcpy(dA, A, sizeof(float) * M * M, hipMemcpyHostToDevice);
    hipMemcpy(dB, B, sizeof(float) * M * M, hipMemcpyHostToDevice);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    dim3 threadPerBlocks(K, K);
    dim3 numBlocks(M/K, M/K);

    hipEventRecord(start);
 
    matrixAdd<<<numBlocks, threadPerBlocks>>>(dA, dB);
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    hipMemcpy(A, dA, sizeof(float) * M * M, hipMemcpyDeviceToHost);
    hipFree(dA);
    hipFree(dB);
    
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    
    if (verbose) {
        printf("Elapsed Time = %f milliseconds\n", elapsedTime);
        verifyMatrixAdd(A, 3.0);
    }
    
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return elapsedTime;
}

int main() {
	int count = 100;
	float averageTime = 0;

	for (int i = 0; i < count; i++)
		averageTime += matrixAddHost(false);

	averageTime /= count;
	printf("[GPU - Float] (Matrix Add 2D) Average Elapsed Time = %f ms\n", averageTime);
	return 0;
}
