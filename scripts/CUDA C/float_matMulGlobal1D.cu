
#include <hip/hip_runtime.h>
#include <stdio.h>

#define M 512
#define P 256
#define N 128
#define K 512

// Implementation of the Matrix Multiplication taken from the Lecture Notes
__global__ void globalMatrixMultiplication1D(float A[M][P], float B[P][N], float C[M][N]) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int i = (int)(idx / N);
    int j = idx % N;
    
    for (int k = 0; k < P; k++)
        C[i][j] += A[i][k] * B[k][j];
}

void verifyMatrixMultiplication(float C[M][N], float check) {
    int maxError = 0;
    
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            maxError += (int)abs(C[i][j] - check);
        }
    }
    
    printf("Maximum Error = %d\n", maxError);
}

float matrixMultiplication1DHost(bool verbose) {
    float A[M][P], B[P][N], C[M][N];
    
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < P; j++)
            A[i][j] = 1.0f;
    }
    
    for (int i = 0; i < P; i++) {
        for (int j = 0; j < N; j++)
            B[i][j] = 2.0f;
    }

    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++)
            C[i][j] = 0.0f;
    }
    
    float (*dA)[P], (*dB)[N], (*dC)[N];
    hipMalloc((void**)&dA, sizeof(float) * M * P);
    hipMalloc((void**)&dB, sizeof(float) * P * N);
    hipMalloc((void**)&dC, sizeof(float) * M * N);
    hipMemcpy(dA, A, sizeof(float) * M * P, hipMemcpyHostToDevice);
    hipMemcpy(dB, B, sizeof(float) * P * N, hipMemcpyHostToDevice);
    hipMemcpy(dC, C, sizeof(float) * M * N, hipMemcpyHostToDevice);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start);

    globalMatrixMultiplication1D<<<M*N/K, K>>>(dA, dB, dC);
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    hipMemcpy(C, dC, sizeof(float) * M * N, hipMemcpyDeviceToHost);
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    
    if (verbose) {
        printf("Elapsed Time = %f milliseconds\n", elapsedTime);
        verifyMatrixMultiplication(C, P * 1.0 * 2.0);
    }
    
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return elapsedTime;
}

int main() {
	int count = 100;
	float averageTime = 0;

	for (int i = 0; i < count; i++)
		averageTime += matrixMultiplication1DHost(false);

	averageTime /= count;
	printf("[GPU - Float] (Matrix Multiplication 1D - Global) Average Elapsed Time = %f ms\n", averageTime);
	return 0;
}
