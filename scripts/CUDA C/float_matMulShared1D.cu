
#include <hip/hip_runtime.h>
#include <stdio.h>

#define M 512
#define P 256
#define N 128
#define K 32

// My own implementation of Matrix Multiplication based on Lecture Notes
__global__ void sharedMatrixMultiplication1D(float A[M][P], float B[P][N], float C[M][N]) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int s_idx = threadIdx.x;
    int i = (int)(idx / N);
    int j = idx % N;
    int s_i = (int)(s_idx / K);
    int s_j = s_idx % K;
    float tempC = 0;
    __shared__ float As[K][K];
    __shared__ float Bs[K][K];
    
    for (int k = 0; k < P / K; k++) {
        As[s_i][s_j] = A[i][k * K + s_j];
        Bs[s_i][s_j] = B[k * K + s_i][j];

        __syncthreads();

        for (int e = 0; e < K; e++)
            tempC += As[s_i][e] * Bs[e][s_j];
    }
    
    C[i][j] = tempC;
}

void verifyMatrixMultiplication(float C[M][N], float check) {
    int maxError = 0;
    
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            maxError += (int)abs(C[i][j] - check);
        }
    }
    
    printf("Maximum Error = %d\n", maxError);
}

float matrixMultiplication1DHost(bool verbose) {
    float A[M][P], B[P][N], C[M][N];
    
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < P; j++)
            A[i][j] = 1.0f;
    }
    
    for (int i = 0; i < P; i++) {
        for (int j = 0; j < N; j++)
            B[i][j] = 2.0f;
    }
    
    float (*dA)[P], (*dB)[N], (*dC)[N];
    hipMalloc((void**)&dA, sizeof(float) * M * P);
    hipMalloc((void**)&dB, sizeof(float) * P * N);
    hipMalloc((void**)&dC, sizeof(float) * M * N);
    hipMemcpy(dA, A, sizeof(float) * M * P, hipMemcpyHostToDevice);
    hipMemcpy(dB, B, sizeof(float) * P * N, hipMemcpyHostToDevice);
    hipMemcpy(dC, C, sizeof(float) * M * N, hipMemcpyHostToDevice);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start);

    sharedMatrixMultiplication1D<<<M*N/(K*K), K*K>>>(dA, dB, dC);
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    hipMemcpy(C, dC, sizeof(float) * M * N, hipMemcpyDeviceToHost);
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    
    if (verbose) {
        printf("Elapsed Time = %f milliseconds\n", elapsedTime);
        verifyMatrixMultiplication(C, P * 1.0 * 2.0);
    }
    
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return elapsedTime;
}

int main() {
	int count = 100;
	float averageTime = 0;

	for (int i = 0; i < count; i++)
		averageTime += matrixMultiplication1DHost(false);

	averageTime /= count;
	printf("[GPU - Float] (Matrix Multiplication 1D - Shared) Average Elapsed Time = %f ms\n", averageTime);
	return 0;
}
