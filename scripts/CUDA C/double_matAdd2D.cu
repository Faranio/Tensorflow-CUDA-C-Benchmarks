
#include <hip/hip_runtime.h>
#include <stdio.h>

#define M 512
#define K 8

// Implementation of the Matrix Add taken from the Lecture Notes
__global__ void matrixAdd(double A[M][M], double B[M][M]) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    A[i][j] += B[i][j];
}

void verifyMatrixAdd(double A[M][M], int check) {
    int maxError = 0;
    
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < M; j++) {
            maxError += (int)abs(A[i][j] - check);
        }
    }
    
    printf("Maximum Error = %d\n", maxError);
}

float matrixAddHost(bool verbose) {
    double A[M][M], B[M][M];
    
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < M; j++) {
            A[i][j] = 1.0;
            B[i][j] = 2.0;
        }
    }
    
    double (*dA)[M], (*dB)[M];
    hipMalloc((void**)&dA, sizeof(double) * M * M);
    hipMalloc((void**)&dB, sizeof(double) * M * M);
    hipMemcpy(dA, A, sizeof(double) * M * M, hipMemcpyHostToDevice);
    hipMemcpy(dB, B, sizeof(double) * M * M, hipMemcpyHostToDevice);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    dim3 threadPerBlocks(K, K);
    dim3 numBlocks(M/K, M/K);

    hipEventRecord(start);
    
    matrixAdd<<<numBlocks, threadPerBlocks>>>(dA, dB);
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    hipMemcpy(A, dA, sizeof(double) * M * M, hipMemcpyDeviceToHost);
    hipFree(dA);
    hipFree(dB);
    
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    
    if (verbose) {
        printf("Elapsed Time = %f milliseconds\n", elapsedTime);
        verifyMatrixAdd(A, 3.0);
    }
    
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return elapsedTime;
}

int main() {
	int count = 100;
	float averageTime = 0;

	for (int i = 0; i < count; i++)
		averageTime += matrixAddHost(false);

	averageTime /= count;
	printf("[GPU - Double] (Matrix Add 2D) Average Elapsed Time = %f ms\n", averageTime);
	return 0;
}
