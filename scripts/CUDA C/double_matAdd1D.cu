
#include <hip/hip_runtime.h>
#include <stdio.h>

#define M 512
#define K 64

// Implementation of Matrix Add from the Lecture Notes
__global__ void matrixAdd1D(double A[M][M], double B[M][M]) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int i = (int)(idx / M);
    int j = idx % M;
    A[i][j] += B[i][j];
}

void verifyMatrixAdd(double A[M][M], int check) {
    int maxError = 0;
    
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < M; j++) {
            maxError += (int)abs(A[i][j] - check);
        }
    }
    
    printf("Maximum Error = %d\n", maxError);
}

float matrixAddHost(bool verbose) {
    double A[M][M], B[M][M];
    
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < M; j++) {
            A[i][j] = 1.0;
            B[i][j] = 2.0;
        }
    }
    
    double (*dA)[M], (*dB)[M];
    hipMalloc((void**)&dA, sizeof(double) * M * M);
    hipMalloc((void**)&dB, sizeof(double) * M * M);
    hipMemcpy(dA, A, sizeof(double) * M * M, hipMemcpyHostToDevice);
    hipMemcpy(dB, B, sizeof(double) * M * M, hipMemcpyHostToDevice);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    
    matrixAdd1D<<<M*M/K, K>>>(dA, dB);
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    hipMemcpy(A, dA, sizeof(double) * M * M, hipMemcpyDeviceToHost);
    hipFree(dA);
    hipFree(dB);
    
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    
    if (verbose) {
        printf("Elapsed Time = %f milliseconds\n", elapsedTime);
        verifyMatrixAdd(A, 3.0);
    }
    
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return elapsedTime;
}

int main() {
	int count = 100;
	float averageTime = 0;

	for (int i = 0; i < count; i++)
		averageTime += matrixAddHost(false);

	averageTime /= count;
	printf("[GPU - Double] (Matrix Add 1D) Average Elapsed Time = %f ms\n", averageTime);
	return 0;
}
